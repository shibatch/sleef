#include "hip/hip_runtime.h"
#include <iostream>
#include <quadmath.h>

#include "sleefquadinline_cuda.h"

// Based on the tutorial code at https://developer.nvidia.com/blog/even-easier-introduction-cuda/

__global__ void pow_gpu(int n, Sleef_quadx1 *r, Sleef_quadx1 *x, Sleef_quadx1 *y) {
  int index = threadIdx.x, stride = blockDim.x;

  for (int i = index; i < n; i += stride)
    r[i] = Sleef_powq1_u10cuda(x[i], y[i]);
}

int main(void) {
  int N = 1 << 20;

  Sleef_quadx1 *rd, *xd, *yd;
  hipMallocManaged(&rd, N*sizeof(Sleef_quadx1));
  hipMallocManaged(&xd, N*sizeof(Sleef_quadx1));
  hipMallocManaged(&yd, N*sizeof(Sleef_quadx1));

  __float128 *r = (__float128 *)rd, *x = (__float128 *)xd, *y = (__float128 *)yd;

  for (int i = 0; i < N; i++) {
    r[i] = 0.0;
    x[i] = 1.00001Q;
    y[i] = i;
  }

  pow_gpu<<<1, 256>>>(N, rd, xd, yd);

  hipDeviceSynchronize();

  double maxError = 0.0;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabsq(r[i]-powq(x[i], y[i])));
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(yd);
  hipFree(xd);
  hipFree(rd);
  
  return 0;
}
